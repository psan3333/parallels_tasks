#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <iostream>
#include <cmath>
#include <vector>
#include <string>
#include <hipcub/hipcub.hpp>

#define CUDACHECK(name) if (hipGetLastError() != hipSuccess || hipDeviceSynchronize() != hipSuccess) throw std::runtime_error(name);

__global__ void interpolate(double* A, double* Anew)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int index = y * blockDim.x + x;
    if (index > blockDim.x && index < blockDim.x * (blockDim.x - 1) - 1) {
        int residual = index % blockDim.x;
        if (residual == 0 || residual == blockDim.x - 1) {
            return;
        }
    }
    else {
        return;
    }

    //average between neighbours
    Anew[index] = 0.25 * (A[index + 1] + A[index - 1] + A[index + blockDim.x] + A[index - blockDim.x]);
}

__global__ void abs_diff(double* A, double* Anew) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int index = y * blockDim.x + x;

    A[index] = A[index] - Anew[index];
    A[index] = A[index] < 0 ? (A[index] * (-1)) : A[index];
}

int main(int argc, char* argv[])
{
    //reads command prompt arguments: ./task4.out [max_aaccuracy] [size] [max_iterations]
    double max_accuracy = std::stod(argv[1]);
    int size = std::stoi(argv[2]);
    int matrixSize = size * size;
    int max_iterations = std::stoi(argv[3]);

    //allocate matrixes and set start conditions (angle values)
    double* A;
    double* Anew;
    hipHostMalloc((void**)&A, matrixSize * sizeof(double));
    CUDACHECK("A host alloc")
    hipHostMalloc((void**)&Anew, matrixSize * sizeof(double));
    CUDACHECK("Anew host alloc")
        
    std::memset(A, 0, matrixSize * sizeof(double));
    std::memset(Anew, 0, matrixSize * sizeof(double));

    A[0] = 10.0;
    A[size - 1] = 20.0;
    A[size * size - 1] = 30.0;
    A[size * (size - 1)] = 20.0;

    Anew[0] = 10.0;
    Anew[size - 1] = 20.0;
    Anew[size * size - 1] = 30.0;
    Anew[size * (size - 1)] = 20.0;
    
    double step = 10.0 / (size - 1);
    for (int i = 1; i < size - 1; i++) {
        A[i] = A[0] + i * step;
        A[i * size] = A[0] + i * step;
        A[size - 1 + size * i] = A[size - 1] + i * step;
        A[size * (size - 1) + i] = A[size * (size - 1)] + i * step;

        Anew[i] = Anew[0] + i * step;
        Anew[i * size] = Anew[0] + i * step;
        Anew[size - 1 + size * i] = Anew[size - 1] + i * step;
        Anew[size * (size - 1) + i] = Anew[size * (size - 1)] + i * step;
    }

    //allocates data on GPU
    double* buff; //buffer for reduciton
    double* dev_A; //GPU copy of matrix A
    double* dev_Anew; //GPU copy of matrix Anew
    hipMalloc((void**)&buff, matrixSize * sizeof(double));
    CUDACHECK("alloc buff");
    hipMalloc((void**)&dev_A, matrixSize * sizeof(double));
    CUDACHECK("alloc dev_A");
    hipMalloc((void**)&dev_Anew, matrixSize * sizeof(double));
    CUDACHECK("alloc dev_Anew");

    //copies values in matrixes 'A' and 'Anew' from CPU to GPU
    hipMemcpy(dev_A, A, matrixSize * sizeof(double), hipMemcpyHostToDevice);
    CUDACHECK("copy from A to dev_A");
    hipMemcpy(dev_Anew, Anew, matrixSize * sizeof(double), hipMemcpyHostToDevice);
    CUDACHECK("copy from Anew to dev_Anew");

    //allocates buffer 'd_out' to contain max('abs_diff' function result)
    double* d_out;
    hipMalloc((void**)&d_out, sizeof(double));
    CUDACHECK("alloc d_out");

    //allocates temporary storage for Max operation and sets temp_storage bytes
    void* d_temp_storage = NULL;
    size_t temp_storage_bytes = 0;
    hipcub::DeviceReduce::Max(d_temp_storage, temp_storage_bytes, buff, d_out, matrixSize);
    hipMalloc(&d_temp_storage, temp_storage_bytes);
    
    size_t threads = (size < 1024) ? size : 1024;
    unsigned int blocks = size / threads;

    dim3 blockDim(threads / 32, threads / 32);
    dim3 gridDim(blocks * 32, blocks * 32);

    double accuracy = max_accuracy + 1.0;
    int num_of_iterations = 0;
    while (num_of_iterations < max_iterations && accuracy > max_accuracy) {

        interpolate<<<gridDim, blockDim>>>(dev_A, dev_Anew);

        //updates accuracy 1/100 times of main cycle iterations
        if (num_of_iterations % 100 == 0 || num_of_iterations + 1 == max_iterations) {

            //fills 'buff' with values from 'dev_A'
            hipMemcpy(buff, dev_A, matrixSize * sizeof(double), hipMemcpyDeviceToDevice);
            CUDACHECK("update dev_A");

            abs_diff<<<gridDim, blockDim>>>(buff, dev_Anew);

            //max reduction
            hipcub::DeviceReduce::Max(d_temp_storage, temp_storage_bytes, buff, d_out, matrixSize);

            hipMemcpy(&accuracy, d_out, sizeof(double), hipMemcpyDeviceToHost);
            CUDACHECK("copy to accuracy");
        }

        ++num_of_iterations;
        std::swap(dev_A, dev_Anew);
    }

    printf("Iterations: %d\nAccuracy: %lf\n", num_of_iterations, accuracy);

    //free memory section
    //GPU free
    hipFree(dev_A);
    CUDACHECK("free dev_A");
    hipFree(dev_Anew);
    CUDACHECK("free dev_Anew");
    hipFree(buff);
    CUDACHECK("free buff");

    //CPU free
    free(A);
    free(Anew);

    return 0;
}
